#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
  
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library 
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
    nvcc -o cuda_passwordcracking cuda_passwordcracking.cu
    ./cuda_passwordcracking
   
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

__device__ int is_a_match(char *attempt) {
  char plain_password1[] = "SA1234";
  char plain_password2[] = "NJ5674";
  char plain_password3[] = "AY2345";
  char plain_password4[] = "TA4567";


  char *x = attempt;
  char *y = attempt;
  char *z = attempt;
  char *a = attempt;
  char *plain1 = plain_password1;
  char *plain2 = plain_password2;
  char *plain3 = plain_password3;
  char *plain4 = plain_password4;

  while(*x == *plain1) { 
   if(*x == '\0') 
    {
	printf("Password found is: %s\n",plain_password1);
      break;
    }

    x++;
    plain1++;
  }
	
  while(*y == *plain2) { 
   if(*y == '\0') 
    {
	printf("Password found is: %s\n",plain_password2);
      break;
    }

    y++;
    plain2++;
  }

  while(*z == *plain3) { 
   if(*z == '\0') 
    {
	printf("Password found is: %s\n",plain_password3);
      break;
    }

    z++;
    plain3++;
  }

  while(*a == *plain4) { 
   if(*a == '\0') 
    {
	printf("Password found is: %s\n",plain_password4);
      return 1;
    }

    a++;
    plain4++;
  }
  return 0;

}


__global__ void  kernel() {
char b,c,d,e;
  
  char password[7];
  password[6] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i; 
char secondValue = j; 
    
password[0] = firstValue;
password[1] = secondValue;
	for(b='0'; b<='9'; b++){
	  for(c='0'; c<='9'; c++){
	   for(d='0'; d<='9'; d++){
	     for(e='0'; e<='9'; e++){
	        password[2] = b;
	        password[3] = c;
	        password[4] = d;
	        password[5] = e; 
	      if(is_a_match(password)) {
		
	      } 
             else {
	     	  
	      }
	   }
	}
	}
	}

}

int time_difference(struct timespec *start, 
                    struct timespec *finish, 
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec; 
  long long int dn =  finish->tv_nsec - start->tv_nsec; 

  if(dn < 0 ) {
    ds--;
    dn += 1000000000; 
  } 
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

  kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

  return 0;
}


